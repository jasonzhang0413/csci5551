#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH       24
#define INSERTION_SORT  32

#define MAXIMUM_VALUE   1000000.0f
#define HANDLE_ERROR( err )  ( HandleError( err, __FILE__, __LINE__ ) )

void HandleError( hipError_t err, const char *file, int line ) {
  //
  // Handle and report on CUDA errors.
  //
  if ( err != hipSuccess ) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );

    exit( EXIT_FAILURE );
  }
}

void checkCUDAError( const char *msg, bool exitOnError ) {
  //
  // Check cuda error and print result if appropriate.
  //
  hipError_t err = hipGetLastError();

  if( hipSuccess != err) {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      if (exitOnError) {
        exit(-1);
      }
  }
}

void cleanupCuda( void ) {
  //
  // Clean up CUDA resources.
  //

  //
  // Explicitly cleans up all runtime-related resources associated with the
  // calling host thread.
  //
  HANDLE_ERROR(
         hipDeviceReset()
         );
}

__device__ float device_pow( float x, float y ) {
  //
  // Calculate x^y on the GPU.
  //
  return pow( x, y );
}

//
// PLACE GPU KERNELS HERE - BEGIN
//

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(float *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        float min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            float val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(float *data, int left, int right, int depth)
{
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    float *lptr = data+left;
    float *rptr = data+right;
    //float  pivot = data[(left+right)/2];
    float  pivot = data[right];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        // Find the next left- and right-hand values to swap
        unsigned int lval = *lptr;
        unsigned int rval = *rptr;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

//
// PLACE GPU KERNELS HERE - END
//

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
void run_qsort(float *data, unsigned int nitems)
{
    // Prepare CDP for the max depth 'MAX_DEPTH'.
    checkCudaErrors(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

    // Launch on device
    int left = 0;
    int right = nitems-1;
    std::cout << "Launching kernel on the GPU" << std::endl;
    cdp_simple_quicksort<<< 1, 1 >>>(data, left, right, 0);
    checkCudaErrors(hipDeviceSynchronize());
}

void printArray(float arr[], int size)
{
	int i;
	printf( "array size is %d\n", size);
	for (i = 0; i < size; i++)
		printf( "%f ", arr[i]);
}

void checkResult(float array[], int size)
{
	float temp = 0;
  bool checkResult = true;
  for (int i=0; i < size; i++) {
    if (temp > array[i]) {
      checkResult = false;
      break;
    }
    temp = array[i];
  }
  if (checkResult) {
    printf( "Result sorted correct\n");
  } else {
    printf( "Result sorted wrong\n");
  }
}

int main( int argc, char* argv[] ) {
  //
  // Determine min, max, mean, mode and standard deviation of array
  //
  unsigned int array_size, seed, i;
  struct timeval start, end;
  double runtime;

  if( argc < 3 ) {
    printf( "Format: quickSort_gpu <size of array> <random seed>\n" );
    printf( "Arguments:\n" );
    printf( "  size of array - This is the size of the array to be generated and processed\n" );
    printf( "  random seed   - This integer will be used to seed the random number\n" );
    printf( "                  generator that will generate the contents of the array\n" );
    printf( "                  to be processed\n" );

    exit( 1 );
  }

  //
  // Get the size of the array to process.
  //
  array_size = atoi( argv[1] );

  //
  // Get the seed to be used
  //
  seed = atoi( argv[2] );

  //
  // Make sure that CUDA resources get cleaned up on exit.
  //
  atexit( cleanupCuda );

  //
  // Record the start time.
  //
  gettimeofday( &start, NULL );

  //
  // Allocate the array to be populated.
  //
  float *array = (float *) malloc( array_size * sizeof( float ) );

  //
  // Seed the random number generator and populate the array with its values.
  //
  srand( seed );
  for( i = 0; i < array_size; i++ )
    array[i] = ( (float) rand() / (float) RAND_MAX ) * MAXIMUM_VALUE;


  float *dev_array;
  // Allocate GPU memory.
  checkCudaErrors(hipMalloc((void **)&dev_array, array_size * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_array, array, array_size * sizeof(float), hipMemcpyHostToDevice));

  // Execute
  run_qsort(dev_array, array_size);

  float *results = new float[array_size];
  checkCudaErrors(hipMemcpy(results, dev_array, array_size*sizeof(float), hipMemcpyDeviceToHost));

  //
  // Record the end time.
  //
  gettimeofday( &end, NULL );

  //
  // Calculate the runtime.
  //
  runtime = ( ( end.tv_sec  - start.tv_sec ) * 1000.0 ) + ( ( end.tv_usec - start.tv_usec ) / 1000.0 );

  //
  // Output discoveries from the array.
  //
  printf( "Statistics for array ( %d, %d ):\n", array_size, seed );
  //printArray(array, array_size);
  printf( "\n------------\n" );
  //printArray(results, array_size);
  checkResult(results, array_size);
  printf( "Processing Time: %4.4f milliseconds\n", runtime );

  //
  // Free the allocated array.
  //
  free( array );
  hipFree(dev_array);

  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset();

  return 0;
}
